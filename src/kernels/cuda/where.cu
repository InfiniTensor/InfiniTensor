#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
#include "cuda/cuda_utility.h"

template <typename T>
__global__ void
_whereKernel(void *inputX, void *inputY, const uint8_t *condition, void *output,
             int a0, int a1, int a2, int a3, int b0, int b1, int b2, int b3,
             int c0, int c1, int c2, int c3, int d0, int d1, int d2, int d3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int outputsize = d0 * d1 * d2 * d3;
    if (i < outputsize) {

        int d0_index = i / (d1 * d2 * d3);
        int d1_index = (i % (d1 * d2 * d3)) / (d2 * d3);
        int d2_index = ((i % (d1 * d2 * d3)) % (d2 * d3)) / d3;
        int d3_index = ((i % (d1 * d2 * d3)) % (d2 * d3)) % d3;

        int a0_index = d0_index % a0;
        int a1_index = d1_index % a1;
        int a2_index = d2_index % a2;
        int a3_index = d3_index % a3;

        int b0_index = d0_index % b0;
        int b1_index = d1_index % b1;
        int b2_index = d2_index % b2;
        int b3_index = d3_index % b3;

        int c0_index = d0_index % c0;
        int c1_index = d1_index % c1;
        int c2_index = d2_index % c2;
        int c3_index = d3_index % c3;

        int inputXIdx = a0_index * a1 * a2 * a3 + a1_index * a2 * a3 +
                        a2_index * a3 + a3_index;
        int inputYIdx = b0_index * b1 * b2 * b3 + b1_index * b2 * b3 +
                        b2_index * b3 + b3_index;
        int conditionIdx = c0_index * c1 * c2 * c3 + c1_index * c2 * c3 +
                           c2_index * c3 + c3_index;
        ((T *)output)[i] = condition[conditionIdx] ? ((T *)inputX)[inputXIdx]
                                                   : ((T *)inputY)[inputYIdx];
    }
}
#define CASE(T)                                                                \
    _whereKernel<DT_CUDA<T>::t>                                                \
        <<<gridsize, blocksize, 0, CUDAStream::stream>>>(                      \
            inputX, inputY, condition, output, a0, a1, a2, a3, b0, b1, b2, b3, \
            c0, c1, c2, c3, d0, d1, d2, d3);

#define SWITCH_DTYPE(DTYPE)                                                    \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASE(1)                                                                \
        break;                                                                 \
    case 2:                                                                    \
        CASE(2)                                                                \
        break;                                                                 \
    case 3:                                                                    \
        CASE(3)                                                                \
        break;                                                                 \
    case 4:                                                                    \
        CASE(4)                                                                \
        break;                                                                 \
    case 5:                                                                    \
        CASE(5)                                                                \
        break;                                                                 \
    case 6:                                                                    \
        CASE(6)                                                                \
        break;                                                                 \
    case 7:                                                                    \
        CASE(7)                                                                \
        break;                                                                 \
    case 10:                                                                   \
        CASE(10)                                                               \
        break;                                                                 \
    case 11:                                                                   \
        CASE(11)                                                               \
        break;                                                                 \
    case 12:                                                                   \
        CASE(12)                                                               \
        break;                                                                 \
    case 13:                                                                   \
        CASE(13)                                                               \
        break;                                                                 \
    case 16:                                                                   \
        CASE(16)                                                               \
        break;                                                                 \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }
namespace infini {

void whereKernel(int dTypeIndex, void *inputX, void *inputY,
                 const uint8_t *condition, void *output, int a0, int a1, int a2,
                 int a3, int b0, int b1, int b2, int b3, int c0, int c1, int c2,
                 int c3, int d0, int d1, int d2, int d3) {
    int blocksize;
    int outputsize = d0 * d1 * d2 * d3;
    if (outputsize > 511) {
        blocksize = 1024;
    } else if (outputsize > 255) {
        blocksize = 512;
    } else if (outputsize > 127) {
        blocksize = 256;
    } else if (outputsize > 63) {
        blocksize = 128;
    } else if (outputsize > 31) {
        blocksize = 64;
    } else {
        blocksize = 32;
    }
    int gridsize = (outputsize + blocksize - 1) / blocksize;

    SWITCH_DTYPE(dTypeIndex)
}

} // namespace infini
