#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
#include <math.h>

constexpr unsigned int num_threads() { return 32 * 4; }
constexpr int thread_work_size() { return 4; }
constexpr int block_work_size() { return thread_work_size() * num_threads(); }

template <class T>
__global__ void _div_kernel(void *x, void *y, void *z, int a0, int a1, int a2,
                            int a3, int b0, int b1, int b2, int b3, int c0,
                            int c1, int c2, int c3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int n = c0 * c1 * c2 * c3;

    for (int i = index; i < n; i += stride) {
        int c0_index = i / (c1 * c2 * c3);
        int c1_index = (i % (c1 * c2 * c3)) / (c2 * c3);
        int c2_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) / c3;
        int c3_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) % c3;

        int a0_index = c0_index % a0;
        int a1_index = c1_index % a1;
        int a2_index = c2_index % a2;
        int a3_index = c3_index % a3;

        int b0_index = c0_index % b0;
        int b1_index = c1_index % b1;
        int b2_index = c2_index % b2;
        int b3_index = c3_index % b3;
        ((T *)z)[i] = ((T *)x)[a0_index * a1 * a2 * a3 + a1_index * a2 * a3 +
                               a2_index * a3 + a3_index] /
                      ((T *)y)[b0_index * b1 * b2 * b3 + b1_index * b2 * b3 +
                               b2_index * b3 + b3_index];
    }
}

template <class T>
__global__ void _add_kernel(void *x, void *y, void *z, int a0, int a1, int a2,
                            int a3, int b0, int b1, int b2, int b3, int c0,
                            int c1, int c2, int c3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int n = c0 * c1 * c2 * c3;

    for (int i = index; i < n; i += stride) {
        int c0_index = i / (c1 * c2 * c3);
        int c1_index = (i % (c1 * c2 * c3)) / (c2 * c3);
        int c2_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) / c3;
        int c3_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) % c3;

        int a0_index = c0_index % a0;
        int a1_index = c1_index % a1;
        int a2_index = c2_index % a2;
        int a3_index = c3_index % a3;

        int b0_index = c0_index % b0;
        int b1_index = c1_index % b1;
        int b2_index = c2_index % b2;
        int b3_index = c3_index % b3;
        ((T *)z)[i] = ((T *)x)[a0_index * a1 * a2 * a3 + a1_index * a2 * a3 +
                               a2_index * a3 + a3_index] +
                      ((T *)y)[b0_index * b1 * b2 * b3 + b1_index * b2 * b3 +
                               b2_index * b3 + b3_index];
    }
}

template <class T>
__global__ void _pow_kernel(void *x, void *y, void *z, int a0, int a1, int a2,
                            int a3, int b0, int b1, int b2, int b3, int c0,
                            int c1, int c2, int c3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int n = c0 * c1 * c2 * c3;

    for (int i = index; i < n; i += stride) {
        int c0_index = i / (c1 * c2 * c3);
        int c1_index = (i % (c1 * c2 * c3)) / (c2 * c3);
        int c2_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) / c3;
        int c3_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) % c3;

        int a0_index = c0_index % a0;
        int a1_index = c1_index % a1;
        int a2_index = c2_index % a2;
        int a3_index = c3_index % a3;

        int b0_index = c0_index % b0;
        int b1_index = c1_index % b1;
        int b2_index = c2_index % b2;
        int b3_index = c3_index % b3;
        ((T *)z)[i] =
            pow(((T *)x)[a0_index * a1 * a2 * a3 + a1_index * a2 * a3 +
                         a2_index * a3 + a3_index],
                ((T *)y)[b0_index * b1 * b2 * b3 + b1_index * b2 * b3 +
                         b2_index * b3 + b3_index]);
    }
}

template <class T>
__global__ void _less_kernel(void *x, void *y, void *z, int a0, int a1, int a2,
                             int a3, int b0, int b1, int b2, int b3, int c0,
                             int c1, int c2, int c3) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int n = c0 * c1 * c2 * c3;

    for (int i = index; i < n; i += stride) {
        int c0_index = i / (c1 * c2 * c3);
        int c1_index = (i % (c1 * c2 * c3)) / (c2 * c3);
        int c2_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) / c3;
        int c3_index = ((i % (c1 * c2 * c3)) % (c2 * c3)) % c3;

        int a0_index = c0_index % a0;
        int a1_index = c1_index % a1;
        int a2_index = c2_index % a2;
        int a3_index = c3_index % a3;

        int b0_index = c0_index % b0;
        int b1_index = c1_index % b1;
        int b2_index = c2_index % b2;
        int b3_index = c3_index % b3;
        ((bool *)z)[i] =
            ((T *)x)[a0_index * a1 * a2 * a3 + a1_index * a2 * a3 +
                     a2_index * a3 + a3_index] <
                    ((T *)y)[b0_index * b1 * b2 * b3 + b1_index * b2 * b3 +
                             b2_index * b3 + b3_index]
                ? true
                : false;
    }
}

namespace infini {
void div_kernel(void *a, void *b, void *c, int a0, int a1, int a2, int a3,
                int b0, int b1, int b2, int b3, int c0, int c1, int c2,
                int c3) {

    int blocksize = block_work_size();
    int num = c0 * c1 * c2 * c3;
    int gridsize = (num + block_work_size() - 1) / block_work_size();
    _div_kernel<float><<<gridsize, blocksize>>>(a, b, c, a0, a1, a2, a3, b0, b1,
                                                b2, b3, c0, c1, c2, c3);
}
void add_kernel(void *a, void *b, void *c, int a0, int a1, int a2, int a3,
                int b0, int b1, int b2, int b3, int c0, int c1, int c2,
                int c3) {

    int blocksize = block_work_size();
    int num = c0 * c1 * c2 * c3;
    int gridsize = (num + block_work_size() - 1) / block_work_size();
    _add_kernel<int64_t><<<gridsize, blocksize>>>(a, b, c, a0, a1, a2, a3, b0,
                                                  b1, b2, b3, c0, c1, c2, c3);
}
void pow_kernel(void *a, void *b, void *c, int a0, int a1, int a2, int a3,
                int b0, int b1, int b2, int b3, int c0, int c1, int c2,
                int c3) {
    int blocksize = block_work_size();
    int num = c0 * c1 * c2 * c3;
    int gridsize = (num + block_work_size() - 1) / block_work_size();
    _pow_kernel<float><<<gridsize, blocksize>>>(a, b, c, a0, a1, a2, a3, b0, b1,
                                                b2, b3, c0, c1, c2, c3);
}
void less_kernel(void *a, void *b, void *c, int a0, int a1, int a2, int a3,
                 int b0, int b1, int b2, int b3, int c0, int c1, int c2,
                 int c3) {
    int blocksize = block_work_size();
    int num = c0 * c1 * c2 * c3;
    int gridsize = (num + block_work_size() - 1) / block_work_size();
    _less_kernel<int64_t><<<gridsize, blocksize>>>(a, b, c, a0, a1, a2, a3, b0,
                                                   b1, b2, b3, c0, c1, c2, c3);
}

}; // namespace infini
