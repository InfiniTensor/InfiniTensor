#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

#define max_function(a, b) ((a) > (b) ? (a) : (b))

template <int BLOCK_DIM_y>
__launch_bounds__(BLOCK_DIM_y) __global__
    void _attentionKernel(const float *__restrict inputQ,
                          const float *__restrict inputK,
                          const float *__restrict inputV, int N, int d,
                          float *__restrict output) {
    int i = blockIdx.x;                              // i must < N,Q[i]
    int phd = threadIdx.y + blockIdx.y * blockDim.y; // V[:,d]

    __shared__ float old_max[BLOCK_DIM_y];
    __shared__ float new_max[BLOCK_DIM_y];
    __shared__ float new_sum[BLOCK_DIM_y];
    old_max[threadIdx.y] = -__FLT_MAX__;
    new_max[threadIdx.y] = -__FLT_MAX__;
    new_sum[threadIdx.y] = 0.0f;

    __shared__ float shareV[BLOCK_DIM_y];
    __shared__ float out[BLOCK_DIM_y];

    int phNumD = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    __shared__ float shareQ_times_K[BLOCK_DIM_y];

    for (int phn = 0; phn < N; phn++) {
        shareV[threadIdx.y] = 0.0f;

        float sum_s = 0.0f;
        for (int ind = 0; ind < phNumD; ind++) {
            if (threadIdx.y + ind * BLOCK_DIM_y < d) {
                shareQ_times_K[threadIdx.y] =
                    inputQ[i * d + threadIdx.y + ind * BLOCK_DIM_y] *
                    inputK[phn * d + threadIdx.y + ind * BLOCK_DIM_y];

            } else {
                shareQ_times_K[threadIdx.y] = 0.0f;
            }
            __syncthreads();
            for (int strip = BLOCK_DIM_y / 2; strip > 0; strip = strip / 2) {
                if (threadIdx.y < strip) {
                    shareQ_times_K[threadIdx.y] +=
                        shareQ_times_K[threadIdx.y + strip];
                }
                __syncthreads();
            }
            sum_s += shareQ_times_K[0];
            __syncthreads();
        }

        shareQ_times_K[threadIdx.y] = sum_s;

        if (phd < d) {
            shareV[threadIdx.y] = inputV[phn * d + phd];
        }

        __syncthreads();

        if (new_max[threadIdx.y] > sum_s) {
            new_sum[threadIdx.y] =
                new_sum[threadIdx.y] + __expf(sum_s - new_max[threadIdx.y]);
        } else {
            new_sum[threadIdx.y] =
                1.0f +
                new_sum[threadIdx.y] * __expf(new_max[threadIdx.y] - sum_s);
            new_max[threadIdx.y] = sum_s;
        }

        __syncthreads();

        shareQ_times_K[threadIdx.y] =
            __expf(shareQ_times_K[threadIdx.y] - new_max[threadIdx.y]);

        __syncthreads();

        if (phn == 0) {
            out[threadIdx.y] =
                shareQ_times_K[threadIdx.y] * shareV[threadIdx.y];

        } else {
            out[threadIdx.y] =
                __expf(old_max[threadIdx.y] - new_max[threadIdx.y]) *
                    out[threadIdx.y] +
                shareQ_times_K[threadIdx.y] * shareV[threadIdx.y];
        }

        old_max[threadIdx.y] = new_max[threadIdx.y];

        __syncthreads();
    }
    __syncthreads();
    if (phd < d)
        output[i * d + phd] =
            out[threadIdx.y] * __fdividef(1.0F, new_sum[threadIdx.y]);
}
namespace infini {
void attentionKernel(const float *inputQ, const float *inputK,
                     const float *inputV, int N, int d, float *output) {

    int num_block_x = N;

    if (d > 1023) {
        int BLOCK_DIM_y = 1024;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<1024>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 511) {
        int BLOCK_DIM_y = 512;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<512>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 255) {
        int BLOCK_DIM_y = 256;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<256>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 127) {
        int BLOCK_DIM_y = 128;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<128>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 63) {
        int BLOCK_DIM_y = 64;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<64>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 31) {
        int BLOCK_DIM_y = 32;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<32>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else {
        int BLOCK_DIM_y = 16;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<16>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    }
}
} // namespace infini
