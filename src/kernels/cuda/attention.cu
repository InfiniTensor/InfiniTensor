#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

template <int BLOCK_DIM_x, int BLOCK_DIM_y>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output) {
    int i = blockIdx.y;                              // i must < N,Q[i]
    int phd = threadIdx.x + blockIdx.x * blockDim.x; // V[:,d]

    int phNumN = (N + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    __shared__ float inputS[BLOCK_DIM_x][BLOCK_DIM_y];
    float newMax;
    float oldMax;
    float newSum;

    newMax = -__FLT_MAX__;
    oldMax = -__FLT_MAX__;
    newSum = 0.0f;

    float out;
    out = 0.0f;
    //---------
    __shared__ float block_sum[BLOCK_DIM_x][BLOCK_DIM_y];

    __shared__ float sum_partial[BLOCK_DIM_x][BLOCK_DIM_y];
    int extra = d % BLOCK_DIM_x;
    int step = (d - extra) / BLOCK_DIM_x;
    for (int phn = 0; phn < phNumN; phn++) {

        int j = threadIdx.y + phn * BLOCK_DIM_y;

        float sum_r = 0.0f;
        __syncthreads();
        if (threadIdx.x < extra) {
            for (int ind = threadIdx.x * (step + 1);
                 ind < (threadIdx.x + 1) * (step + 1); ind++) {
                sum_r += inputQ[i * d + ind] * inputK[j * d + ind];
            }
        } else {
            for (int ind = extra * (step + 1) + (threadIdx.x - extra) * step;
                 ind < extra * (step + 1) + (threadIdx.x - extra + 1) * step;
                 ind++) {
                sum_r += inputQ[i * d + ind] * inputK[j * d + ind];
            }
        }
        if (j < N) {
            sum_partial[threadIdx.x][threadIdx.y] = sum_r;
        } else {
            sum_partial[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = BLOCK_DIM_x / 2; strip > 0; strip /= 2) {
            if (threadIdx.x < strip) {
                sum_partial[threadIdx.x][threadIdx.y] +=
                    sum_partial[threadIdx.x + strip][threadIdx.y];
            }
            __syncthreads();
        }
        float sum_s = sum_partial[0][threadIdx.y];
        if (j < N) {

            block_sum[threadIdx.x][threadIdx.y] = 1.0f;
        } else {

            sum_partial[0][threadIdx.y] = -__FLT_MAX__;
            block_sum[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = BLOCK_DIM_y / 2; strip > 0; strip /= 2) {
            if (threadIdx.y < strip) {
                if (sum_partial[0][threadIdx.y] >
                    sum_partial[0][threadIdx.y + strip]) {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x][threadIdx.y] +
                        block_sum[threadIdx.x][threadIdx.y + strip] *
                            __expf(sum_partial[0][threadIdx.y + strip] -
                                   sum_partial[0][threadIdx.y]);
                } else {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x][threadIdx.y + strip] +
                        block_sum[threadIdx.x][threadIdx.y] *
                            __expf(sum_partial[0][threadIdx.y] -
                                   sum_partial[0][threadIdx.y + strip]);
                    sum_partial[0][threadIdx.y] =
                        sum_partial[0][threadIdx.y + strip];
                }
            }
            __syncthreads();
        }
        if (newMax > sum_partial[0][0]) {
            newSum = newSum + block_sum[threadIdx.x][0] *
                                  __expf(sum_partial[0][0] - newMax);
        } else {
            newSum = block_sum[threadIdx.x][0] +
                     newSum * __expf(newMax - sum_partial[0][0]);
            newMax = sum_partial[0][0];
        }

        if (j < N && phd < d) {
            inputS[threadIdx.x][threadIdx.y] =
                __expf(sum_s - newMax) *
                inputV[(threadIdx.y + phn * BLOCK_DIM_y) * d + phd];
        } else {
            inputS[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = BLOCK_DIM_y / 2; strip > 0; strip /= 2) {
            if (threadIdx.y < strip) {
                inputS[threadIdx.x][threadIdx.y] +=
                    inputS[threadIdx.x][threadIdx.y + strip];
            }
            __syncthreads();
        }
        if (j < N && phd < d) {
            out = __expf(oldMax - newMax) * out + inputS[threadIdx.x][0];
        }
        oldMax = newMax;
    }

    if (threadIdx.y + (phNumN - 1) * BLOCK_DIM_y < N && phd < d) {
        output[i * d + phd] = out * __fdividef(1.0F, newSum);
    }
}
namespace infini {
void attentionKernel(const float *inputQ, const float *inputK,
                     const float *inputV, int N, int d, float *output) {
    int num_block_y = N;
    if (d > 512) {
        int BLOCK_DIM_x = 1024;
        int BLOCK_DIM_y = 1;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<1024, 1>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 256) {
        int BLOCK_DIM_x = 512;
        int BLOCK_DIM_y = 2;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<512, 2>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 128) {
        int BLOCK_DIM_x = 256;
        int BLOCK_DIM_y = 4;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<256, 4>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 64) {
        int BLOCK_DIM_x = 128;
        int BLOCK_DIM_y = 8;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<128, 8>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 32) {
        int BLOCK_DIM_x = 64;
        int BLOCK_DIM_y = 16;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<64, 16>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 16) {
        int BLOCK_DIM_x = 32;
        int BLOCK_DIM_y = 32;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<32, 32>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else {
        int BLOCK_DIM_x = 16;
        int BLOCK_DIM_y = 64;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<16, 64>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    }
}
} // namespace infini
