#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

#define max_function(a, b) ((a) > (b) ? (a) : (b))

template <int BLOCK_DIM_y>
__launch_bounds__(BLOCK_DIM_y) __global__
    void _attentionKernel(const float *__restrict inputQ,
                          const float *__restrict inputK,
                          const float *__restrict inputV, int N, int d,
                          float *__restrict output) {
    int i = blockIdx.x;                              // i must < N,Q[i]
    int phd = threadIdx.y + blockIdx.y * blockDim.y; // V[:,d]

    __shared__ float old_max[BLOCK_DIM_y];
    __shared__ float new_max[BLOCK_DIM_y];
    __shared__ float new_sum[BLOCK_DIM_y];
    old_max[threadIdx.y] = -__FLT_MAX__;
    new_max[threadIdx.y] = -__FLT_MAX__;
    new_sum[threadIdx.y] = 0.0f;
    __shared__ float block_sum[BLOCK_DIM_y];
    __shared__ float block_max[BLOCK_DIM_y];

    __shared__ float inputS[BLOCK_DIM_y];
    __shared__ float shareV[BLOCK_DIM_y];
    __shared__ float out[BLOCK_DIM_y];

    int phNumD = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    __shared__ float shareQ[BLOCK_DIM_y];
    __shared__ float shareK[BLOCK_DIM_y];
    for (int phn = 0; phn < N; phn++) {
        shareV[threadIdx.y] = 0.0f;

        float sum_s = 0;

        for (int ind = 0; ind < phNumD; ind++) {
            if (threadIdx.y + ind * BLOCK_DIM_y < d) {
                shareQ[threadIdx.y] =
                    inputQ[i * d + threadIdx.y + ind * BLOCK_DIM_y];
                shareK[threadIdx.y] =
                    inputK[phn * d + threadIdx.y + ind * BLOCK_DIM_y];
            } else {
                shareQ[threadIdx.y] = 0.0f;
                shareK[threadIdx.y] = 0.0f;
            }
            __syncthreads();
            for (int index = 0; index < BLOCK_DIM_y; index++) {
                sum_s += shareQ[index] * shareK[index];
            }
            __syncthreads();
        }

        inputS[threadIdx.y] = sum_s;
        block_max[threadIdx.y] = sum_s;
        block_sum[threadIdx.y] = 1.0f;

        if (phd < d) {
            shareV[threadIdx.y] = inputV[phn * d + phd];
        }

        __syncthreads();

        if (new_max[threadIdx.y] > block_max[threadIdx.y]) {
            new_sum[threadIdx.y] =
                new_sum[threadIdx.y] +
                block_sum[threadIdx.y] *
                    __expf(block_max[threadIdx.y] - new_max[threadIdx.y]);
        } else {
            new_sum[threadIdx.y] =
                block_sum[threadIdx.y] +
                new_sum[threadIdx.y] *
                    __expf(new_max[threadIdx.y] - block_max[threadIdx.y]);
            new_max[threadIdx.y] = block_max[threadIdx.y];
        }

        __syncthreads();

        inputS[threadIdx.y] =
            __expf(inputS[threadIdx.y] - new_max[threadIdx.y]);

        __syncthreads();

        if (phn == 0) {
            out[threadIdx.y] = inputS[threadIdx.y] * shareV[threadIdx.y];

        } else {
            out[threadIdx.y] =
                __expf(old_max[threadIdx.y] - new_max[threadIdx.y]) *
                    out[threadIdx.y] +
                inputS[threadIdx.y] * shareV[threadIdx.y];
        }

        old_max[threadIdx.y] = new_max[threadIdx.y];

        __syncthreads();
    }
    __syncthreads();
    if (phd < d)
        output[i * d + phd] =
            out[threadIdx.y] * __fdividef(1.0F, new_sum[threadIdx.y]);
}
namespace infini {
void attentionKernel(const float *inputQ, const float *inputK,
                     const float *inputV, int N, int d, float *output) {

    int num_block_x = N;

    if (d > 1023) {
        int BLOCK_DIM_y = 1024;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<1024>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 511) {
        int BLOCK_DIM_y = 512;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<512>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 255) {
        int BLOCK_DIM_y = 256;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<256>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 127) {
        int BLOCK_DIM_y = 128;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<128>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 63) {
        int BLOCK_DIM_y = 64;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<64>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 31) {
        int BLOCK_DIM_y = 32;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<32>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else {
        int BLOCK_DIM_y = 16;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<16>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    }
}
} // namespace infini
