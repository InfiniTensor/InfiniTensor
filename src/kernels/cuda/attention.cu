#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
const int Rq = 4;
const int Rv = 8; // 必须是4的倍数
const int Br = 16;
const int Bc = 16;
const int Bk = 4; // 必须是4的倍数

template <int Br, int Bc, int Rq>
__device__ void matmulRQK(const float *__restrict inputQ,
                          const float *__restrict inputK, float *shareQK,
                          float *shareVK, int N, int d, int width, int indQ,
                          int indK, float *val) {
    float a[4];
    for (int ph = 0; ph < width; ph++) {
        for (int index_k = 0; index_k < Bk; index_k++) {
            (float4 &)a[0] = (float4 &)
                inputK[(indK + index_k) * d + (threadIdx.y + ph * Bc) * Bk];
            for (int idk = 0; idk < Bk; idk++) {
                if (threadIdx.y < Bc) {
                    shareVK[(threadIdx.y * Bk + idk) * Bc * Bk +
                            threadIdx.x * Bk + index_k] = a[idk];
                    if (indK + index_k >= N ||
                        (threadIdx.y + ph * Bc) * Bk + idk >= d) {

                        shareVK[(threadIdx.y * Bk + idk) * Bc * Bk +
                                threadIdx.x * Bk + index_k] = 0.0f;
                    }
                }
            }
        }

        for (int index_q = 0; index_q < Rq; index_q++) {
            (float4 &)shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                              threadIdx.x * Bk] = (float4 &)
                inputQ[(indQ + index_q) * d + (threadIdx.x + ph * Bc) * Bk];
            for (int idk = 0; idk < Bk; idk++) {
                if (indQ + index_q >= N ||
                    (threadIdx.x + ph * Bc) * Bk + idk >= d) {
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                            threadIdx.x * Bk + idk] = 0.0f;
                }
            }
        }
        __syncthreads();

        for (int index = 0; index < Bc * Bk; index++) {
            for (int index_q = 0; index_q < Rq; index_q++) {
                for (int index_k = 0; index_k < Bk; index_k++) {
                    val[index_q * Bk + index_k] = std::fma(
                        shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk + index],
                        shareVK[index * Bc * Bk + threadIdx.x * Bk + index_k],
                        val[index_q * Bk + index_k]);
                }
            }
        }
        __syncthreads();
    }
}
template <int Br, int Bc, int Rq, int Rv>
__device__ void matmulSV(float *shareQK, const float *__restrict inputV,
                         float *shareVK, int N, int d, int j, int indQ,
                         int indK, int indV, float *val, float *newMax,
                         float *sumSV) {
    if (threadIdx.y < Bc) {
        for (int index_k = 0; index_k < Bk; index_k++) {
            for (int id = 0; id < (int)(Rv / 4); id++) {
                (float4 &)shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                                  threadIdx.x * Rv + id * 4] = (float4 &)
                    inputV[((threadIdx.y + j * Bc) * Bk + index_k) * d + indV +
                           id * 4];
            }
            for (int index_v = 0; index_v < Rv; index_v++) {
                if ((threadIdx.y + j * Bc) * Bk + index_k >= N ||
                    indV + index_v >= d) {
                    shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                            threadIdx.x * Rv + index_v] = 0.0f;
                }
            }
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++) {
        for (int index_k = 0; index_k < Bk; index_k++) {
            if (indQ + index_q < N && indK + index_k < N) {
                shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                        threadIdx.x * Bk + index_k] =
                    __expf(val[index_q * Bk + index_k] - newMax[index_q]);
            } else {

                shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                        threadIdx.x * Bk + index_k] = 0.0f;
            }
        }
    }
    __syncthreads();

    for (int phc = 0; phc < Bc * Bk; phc++) {
        for (int index_q = 0; index_q < Rq; index_q++) {

            for (int index_v = 0; index_v < Rv; index_v++) {
                sumSV[index_q * Rv + index_v] +=
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk + phc] *
                    shareVK[phc * Bc * Rv + threadIdx.x * Rv + index_v];
            }
        }
    }
}
template <typename T> struct SumOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

template <typename T> struct MaxOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = 32>
__inline__ __device__ T WarpAllReduce(T val) {
    for (int mask = thread_group_width / 2; mask > 0; mask >>= 1) {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }

    return val;
}

template <int Br, int Bc, int Rq, int Rv>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output) {

    __shared__ float shareQK[Rq * Br * Bc * Bk];
    __shared__ float shareVK[Bk * Bc * Bc * Rv];

    float sumSV[Rq * Rv] = {0.0f};
    float newMax[Rq];
    float oldMax[Rq];
    float newSum[Rq] = {0.0f};

    float val[Rq * Bk];

    int indV = Rv * (threadIdx.x + blockIdx.x * blockDim.x);
    int indQ = Rq * (threadIdx.y + blockIdx.y * blockDim.y);

    for (int index_q = 0; index_q < Rq; index_q++) {
        newMax[index_q] = -__FLT_MAX__;
        oldMax[index_q] = -__FLT_MAX__;
    }

    int Tc = (N + Bc * Bk - 1) / (Bc * Bk);

    int width = (d + Bc * Bk - 1) / (Bc * Bk);
    for (int j = 0; j < Tc; j++) {

        int indK = Bk * (threadIdx.x + j * Bc);
        for (int index_q = 0; index_q < Rq; index_q++) {
            for (int index_k = 0; index_k < Bk; index_k++) {

                val[index_q * Bk + index_k] = 0.0f;
            }
        }
        matmulRQK<Br, Bc, Rq>(inputQ, inputK, shareQK, shareVK, N, d, width,
                              indQ, indK, val);
        for (int index_q = 0; index_q < Rq; index_q++) {
            float tmpReduceMax = -__FLT_MAX__;
            for (int index_k = 0; index_k < Bk; index_k++) {
                if (indQ + index_q < N && indK + index_k < N) {

                    tmpReduceMax =
                        max(tmpReduceMax, val[index_q * Bk + index_k]);
                }
            }
            __syncthreads();
            tmpReduceMax = WarpAllReduce<MaxOp, float, Bc>(tmpReduceMax);
            if (threadIdx.x == 0) {
                shareQK[threadIdx.y * Rq + index_q] = tmpReduceMax;
            }
            __syncthreads();
            float tmpReduceSum = 0.0f;
            for (int index_k = 0; index_k < Bk; index_k++) {
                if (indQ + index_q < N && indK + index_k < N) {
                    tmpReduceSum += __expf(val[index_q * Bk + index_k] -
                                           shareQK[threadIdx.y * Rq + index_q]);
                }
            }
            __syncthreads();
            tmpReduceSum = WarpAllReduce<SumOp, float, Bc>(tmpReduceSum);
            if (threadIdx.x == 0) {
                shareQK[threadIdx.y * Rq + index_q + Rq * Br] = tmpReduceSum;
            }
            __syncthreads();
            if (newMax[index_q] > shareQK[threadIdx.y * Rq + index_q]) {
                newSum[index_q] =
                    std::fma(shareQK[threadIdx.y * Rq + index_q + Rq * Br],
                             __expf(shareQK[threadIdx.y * Rq + index_q] -
                                    newMax[index_q]),
                             newSum[index_q]);
            } else {
                newSum[index_q] =
                    std::fma(newSum[index_q],
                             __expf(newMax[index_q] -
                                    shareQK[threadIdx.y * Rq + index_q]),
                             shareQK[threadIdx.y * Rq + index_q + Rq * Br]);

                newMax[index_q] = shareQK[threadIdx.y * Rq + index_q];
            }
            // PV
            for (int index_v = 0; index_v < Rv; index_v++) {
                sumSV[index_q * Rv + index_v] *=
                    __expf(oldMax[index_q] - newMax[index_q]);
            }
        }

        matmulSV<Br, Bc, Rq, Rv>(shareQK, inputV, shareVK, N, d, j, indQ, indK,
                                 indV, val, newMax, sumSV);

        for (int index_q = 0; index_q < Rq; index_q++) {
            oldMax[index_q] = newMax[index_q];
        }

        //__syncthreads();
    }
    for (int index_q = 0; index_q < Rq; index_q++) {
        float inv = __fdividef(1.0F, newSum[index_q]);
        for (int index_v = 0; index_v < Rv; index_v++) {
            sumSV[index_q * Rv + index_v] = sumSV[index_q * Rv + index_v] * inv;
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++) {

        for (int id = 0; id < (int)(Rv / 4); id++) {
            if (indQ + index_q < N) {
                (float4 &)output[(indQ + index_q) * d + indV + id * 4] =
                    (float4 &)sumSV[index_q * Rv + id * 4];
            }
        }
    }
}
namespace infini {
void attentionKernel(const float *inputQ, const float *inputK,
                     const float *inputV, int N, int d, float *output) {
    int num_block_x = (d + Rv * Bc - 1) / (Rv * Bc);
    int num_block_y = (N + Rq * Br - 1) / (Rq * Br);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(Bc, Br, 1);

    _attentionKernel<Br, Bc, Rq, Rv>
        <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
}
} // namespace infini
