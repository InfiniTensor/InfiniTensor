#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
#include "utils/small_array.h"
#include <hipcub/hipcub.hpp>

struct __align__(8) MD { // update the global max and sum, store the output at
                         // max_tmp and sum_tmp
    float max_tmp;       // store max
    float sum_tmp;       // store sum
};
__device__ __forceinline__ MD reduce_md_op(MD a, MD b) {
    bool a_bigger = (a.max_tmp > b.max_tmp);
    MD bigger = a_bigger ? a : b;
    MD smaller = a_bigger ? b : a;
    MD res;
    res.sum_tmp = bigger.sum_tmp +
                  smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);
    res.max_tmp = bigger.max_tmp;
    return res;
}
template <int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__
    void _softmax_kernel(float *input, float *output, int size,
                         infini::SmallArray inputShape, int axis, int nDims,
                         int stride) { // if set axis = 1, inputShape=[I,J,K,S]
    int tid = 0;                       // tid = i(JKS) + j(KS) + k(S) + s
    int dimsize = inputShape.data[axis]; // set axis = 1, dimsize = J
    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    tid = blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                    dimsize; // now, tid = i(JKS) + k(S) + s;

    MD md_partial;
    md_partial.max_tmp = -__FLT_MAX__;
    md_partial.sum_tmp = 0.0f;
    for (int id = threadIdx.x; id < dimsize; id += blockDim.x) {
        MD md_input;
        md_input.max_tmp = input[tid + id * stride];
        md_input.sum_tmp = 1.0f;
        md_partial = reduce_md_op(md_partial,
                                  md_input); // reduce the data to one block
    }
    typedef hipcub::BlockReduce<MD, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ MD md_total;
    MD md_block = BlockReduce(temp_storage).Reduce(md_partial, reduce_md_op);
    if (threadIdx.x ==
        0) { // must set threadIdx.x = 0 write the output to memory
        md_total = md_block;
    }
    __syncthreads();
    //-----------------
    float max_total, sum_inverse_total;
    max_total = md_total.max_tmp;
    sum_inverse_total = __fdividef(1.0F, md_total.sum_tmp);
    for (int id = threadIdx.x; id < dimsize; id += blockDim.x) {
        output[tid + id * stride] =
            __expf(input[tid + id * stride] - max_total) * sum_inverse_total;
    }
}
namespace infini {
void softmax_kernel(float *input, float *output, int size,
                    SmallArray inputShape, int axis, int nDims, int stride) {
    int dimsize = inputShape.data[axis];
    int num_blocks = size / dimsize;
    if (dimsize > 1023) {
        int BLOCK_DIM = 1024;
        _softmax_kernel<1024><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    } else if (dimsize > 511) {
        int BLOCK_DIM = 512;
        _softmax_kernel<512><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    } else if (dimsize > 255) {
        int BLOCK_DIM = 256;
        _softmax_kernel<256><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    } else if (dimsize > 127) {
        int BLOCK_DIM = 128;
        _softmax_kernel<128><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    } else if (dimsize > 63) {
        int BLOCK_DIM = 64;
        _softmax_kernel<64><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    } else {
        int BLOCK_DIM = 32;
        _softmax_kernel<32><<<num_blocks, BLOCK_DIM>>>(
            input, output, size, inputShape, axis, nDims, stride);
    }
}
} // namespace infini
