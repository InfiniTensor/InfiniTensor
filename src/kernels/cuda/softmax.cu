#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

#include <hipcub/hipcub.hpp>

struct __align__(8) MD { // update the global max and sum, store the output at
                         // max_tmp and sum_tmp
    float max_tmp;       // store max
    float sum_tmp;       // store sum
};
__device__ __forceinline__ MD reduce_md_op(MD a, MD b) {
    bool a_bigger = (a.max_tmp > b.max_tmp);
    MD bigger = a_bigger ? a : b;
    MD smaller = a_bigger ? b : a;
    MD res;
    res.sum_tmp = bigger.sum_tmp +
                  smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);
    res.max_tmp = bigger.max_tmp;
    return res;
}
template <int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__
    void _softmax_kernel(float *__restrict input, float *__restrict output,
                         int size, int dimsize,
                         int stride) { // if set axis = 1, inputShape=[I,J,K,S]
    int tid = 0;                       // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    tid = blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                    dimsize; // now, tid = i(JKS) + k(S) + s;
    __shared__ float share_input[BLOCK_DIM];
    share_input[threadIdx.x] = input[tid + threadIdx.x * stride];
    __syncthreads();
    MD md_partial;
    md_partial.max_tmp = -__FLT_MAX__;
    md_partial.sum_tmp = 0.0f;
    for (int id = threadIdx.x; id < dimsize; id += blockDim.x) {
        MD md_input;
        if (id < BLOCK_DIM) {
            md_input.max_tmp = share_input[id];
        } else {
            md_input.max_tmp = input[tid + id * stride];
        }

        md_input.sum_tmp = 1.0f;
        md_partial = reduce_md_op(md_partial,
                                  md_input); // reduce the data to one block
    }
    typedef hipcub::BlockReduce<MD, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ MD md_total;
    MD md_block = BlockReduce(temp_storage).Reduce(md_partial, reduce_md_op);
    if (threadIdx.x ==
        0) { // must set threadIdx.x = 0 write the output to memory
        md_total = md_block;
    }
    __syncthreads();
    //-----------------
    float max_total, sum_inverse_total;
    max_total = md_total.max_tmp;
    sum_inverse_total = __fdividef(1.0F, md_total.sum_tmp);
    for (int id = threadIdx.x; id < dimsize; id += blockDim.x) {
        if (id < BLOCK_DIM) {
            output[tid + id * stride] =
                __expf(share_input[id] - max_total) * sum_inverse_total;
        } else {
            output[tid + id * stride] =
                __expf(input[tid + id * stride] - max_total) *
                sum_inverse_total;
        }
    }
}
namespace infini {
void softmax_kernel(int num_blocks, float *input, float *output, int size,
                    int dimsize, int stride) {
    if (dimsize > 1023) {
        int BLOCK_DIM = 1024;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<1024><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 511) {
        int BLOCK_DIM = 512;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<512><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 255) {
        int BLOCK_DIM = 256;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<256><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 127) {
        int BLOCK_DIM = 128;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<128><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 63) {
        int BLOCK_DIM = 64;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<64><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else {
        int BLOCK_DIM = 32;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<32><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    }
}
} // namespace infini