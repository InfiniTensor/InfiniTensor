#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
#include <hipcub/hipcub.hpp>

struct __align__(8) DataMaxSum { // update the global max and sum, store the
                                 // output at max_tmp and sum_tmp
    float max_tmp;               // store max
    float sum_tmp;               // store sum
};
__device__ __forceinline__ DataMaxSum reduce_dms_op(DataMaxSum a,
                                                    DataMaxSum b) {
    bool a_bigger = (a.max_tmp > b.max_tmp);
    DataMaxSum bigger = a_bigger ? a : b;
    DataMaxSum smaller = a_bigger ? b : a;
    bigger.sum_tmp = bigger.sum_tmp +
                     smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);

    return bigger;
}
template <int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__ void _blockSoftmaxKernel(
    float *__restrict input, float *__restrict output, int size, int dimsize,
    int stride) { // if set axis = 1, inputShape=[I,J,K,S]
                  // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    int tid =
        blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                  dimsize; // now, tid = i(JKS) + k(S) + s;

    DataMaxSum dms_partial;
    dms_partial.max_tmp = -__FLT_MAX__;
    dms_partial.sum_tmp = 0.0f;
    DataMaxSum dms_input;
    for (int ph = 0; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++) {

        dms_input.max_tmp =
            input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride];

        dms_input.sum_tmp = 1.0f;
        dms_partial = reduce_dms_op(dms_partial,
                                    dms_input); // reduce the data to one block
    }
    typedef hipcub::BlockReduce<DataMaxSum, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ DataMaxSum dms_total;
    DataMaxSum dms_block =
        BlockReduce(temp_storage).Reduce(dms_partial, reduce_dms_op);
    if (threadIdx.x ==
        0) { // must set threadIdx.x = 0 write the output to memory
        dms_total = dms_block;
    }
    __syncthreads();
    //-----------------

    for (int ph = 0; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++) {
        output[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] =
            __expf(input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] -
                   dms_total.max_tmp) *
            __fdividef(1.0F, dms_total.sum_tmp);
    }
}

template <typename T> struct SumOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

template <typename T> struct MaxOp {
    __device__ __forceinline__ T operator()(const T &a, const T &b) const {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width>
__inline__ __device__ T WarpAllReduce(T val) {
    for (int mask = thread_group_width / 2; mask > 0; mask /= 2) {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}
template <int BLOCK_DIM_x, int BLOCK_DIM_y>
__global__ void _warpSoftmaxKernel(float *__restrict input,
                                   float *__restrict output, int size,
                                   int dimsize, int stride) {
    int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
    int otherSize = size / dimsize;
    int tid = otherIdx % stride + (otherIdx - otherIdx % stride) * dimsize;

    if (otherIdx < otherSize) {

        __shared__ float max_total[BLOCK_DIM_y];
        __shared__ float sum_total[BLOCK_DIM_y];
        float max_data = -__FLT_MAX__;

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++) {
            max_data =
                max(max_data,
                    input[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride]);
        }

        max_data = WarpAllReduce<MaxOp, float, BLOCK_DIM_x>(max_data);

        if (threadIdx.x == 0)
            max_total[threadIdx.y] = max_data;

        //--------------------------------------------
        float sum_data = 0.0f;

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++) {
            sum_data +=
                __expf(input[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride] -
                       max_total[threadIdx.y]);
        }

        sum_data = WarpAllReduce<SumOp, float, BLOCK_DIM_x>(sum_data);

        if (threadIdx.x == 0)
            sum_total[threadIdx.y] = sum_data;

        //--------------------------------------------

        for (int ph = 0; threadIdx.x + ph * BLOCK_DIM_x < dimsize; ph++) {
            output[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride] =
                __expf(input[tid + (threadIdx.x + ph * BLOCK_DIM_x) * stride] -
                       max_total[threadIdx.y]) *
                __fdividef(1.0F, sum_total[threadIdx.y]);
        }
    }
}
//-----------------

//-----------------
namespace infini {
void softmax_kernel(int num_blocks, float *input, float *output, int size,
                    int dimsize, int stride) {

    if (dimsize > 1024) {

        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<1024>
            <<<num_blocks, BLOCK_DIM>>>(input, output, size, dimsize, stride);
    } else if (dimsize > 31) {
        int BLOCK_DIM_x = 32;
        int BLOCK_DIM_y = 32;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<32, 32>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    } else if (dimsize > 15) {
        int BLOCK_DIM_x = 16;
        int BLOCK_DIM_y = 64;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<16, 64>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    } else if (dimsize > 7) {
        int BLOCK_DIM_x = 8;
        int BLOCK_DIM_y = 128;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<8, 128>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    } else {
        int BLOCK_DIM_x = 4;
        int BLOCK_DIM_y = 256;
        int num_block_x = (num_blocks + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<4, 256>
            <<<grid_dim, block_dim>>>(input, output, size, dimsize, stride);
    }
}
} // namespace infini
