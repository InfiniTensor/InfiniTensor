#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

#include <hipcub/hipcub.hpp>

struct __align__(8) MD { // update the global max and sum, store the output at
                         // max_tmp and sum_tmp
    float max_tmp;       // store max
    float sum_tmp;       // store sum
};
__device__ __forceinline__ MD reduce_md_op(MD a, MD b) {
    bool a_bigger = (a.max_tmp > b.max_tmp);
    MD bigger = a_bigger ? a : b;
    MD smaller = a_bigger ? b : a;
    MD res;
    res.sum_tmp = bigger.sum_tmp +
                  smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);
    res.max_tmp = bigger.max_tmp;
    return res;
}
template <int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__
    void _softmax_kernel(float *__restrict input, float *__restrict output,
                         int size, int dimsize,
                         int stride) { // if set axis = 1, inputShape=[I,J,K,S]
                                       // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    int tid =
        blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                  dimsize; // now, tid = i(JKS) + k(S) + s;
    __shared__ float share_input[BLOCK_DIM];
    if (threadIdx.x < dimsize) {
        share_input[threadIdx.x] = input[tid + threadIdx.x * stride];
    } else {
        share_input[threadIdx.x] = -__FLT_MAX__;
    }
    __syncthreads();
    MD md_partial;
    md_partial.max_tmp = share_input[threadIdx.x];
    md_partial.sum_tmp = 1.0f;
    MD md_input;
    for (int ph = 1; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++) {

        md_input.max_tmp = input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride];

        md_input.sum_tmp = 1.0f;
        md_partial = reduce_md_op(md_partial,
                                  md_input); // reduce the data to one block
    }
    typedef hipcub::BlockReduce<MD, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ MD md_total;
    MD md_block = BlockReduce(temp_storage).Reduce(md_partial, reduce_md_op);
    if (threadIdx.x ==
        0) { // must set threadIdx.x = 0 write the output to memory
        md_total = md_block;
    }
    __syncthreads();
    //-----------------
    float max_total, sum_inverse_total;
    max_total = md_total.max_tmp;
    sum_inverse_total = __fdividef(1.0F, md_total.sum_tmp);
    if (threadIdx.x < dimsize) {
        output[tid + (threadIdx.x) * stride] =
            __expf(share_input[threadIdx.x] - max_total) * sum_inverse_total;
    }

    for (int ph = 1; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++) {
        output[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] =
            __expf(input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] -
                   max_total) *
            sum_inverse_total;
    }
}
namespace infini {
void softmax_kernel(int num_blocks, float *input, float *output, int size,
                    int dimsize, int stride) {
    if (dimsize > 1023) {
        int BLOCK_DIM = 1024;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<1024><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 511) {
        int BLOCK_DIM = 512;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<512><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 255) {
        int BLOCK_DIM = 256;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<256><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 127) {
        int BLOCK_DIM = 128;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<128><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 63) {
        int BLOCK_DIM = 64;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<64><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else if (dimsize > 31) {
        int BLOCK_DIM = 32;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<32><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    } else {
        int BLOCK_DIM = 16;
        int share_mem = BLOCK_DIM * sizeof(float);
        _softmax_kernel<16><<<num_blocks, BLOCK_DIM, share_mem>>>(
            input, output, size, dimsize, stride);
    }
}
} // namespace infini