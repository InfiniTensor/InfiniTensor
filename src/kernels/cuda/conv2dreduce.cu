#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"

__global__ void
conv2dreduce_kernel_(float *__restrict__ input, float *__restrict__ bias,
                     float *__restrict__ output, const bool PReLU,
                     const float paramReLU, const int n, const int f,
                     const int h, const int w, const int oh, const int ow,
                     const int r, const int s, const int ph, const int pw,
                     const int dh, const int dw, const int sh, const int sw) {
    // output shape: (n, oh, ow, f)
    // input shape: (n, h, w, f, r, s)
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int out_N_offset = h * w * f, out_H_offset = w * f, out_W_offset = f,
              out_F_offset = 1;
    const int num = out_N_offset * n;
    if (tid < num) {
        // output index
        int tmptid = tid;
        const int nid = tmptid / out_N_offset;
        tmptid -= nid * out_N_offset;
        const int hid = tmptid / out_H_offset;
        tmptid -= hid * out_H_offset;
        const int wid = tmptid / out_W_offset;
        tmptid -= wid * out_W_offset;
        const int fid = tmptid / out_F_offset;

        // Input index
        const int fchunck = r * s, wchunk = f * fchunck, hchunk = w * wchunk,
                  nchunck = n * hchunk;
        float *__restrict__ nfinput = input + nid * nchunck + fid * fchunck;
        float imm = 0.0;
        const int ihst = hid * sh, iwst = wid * sw;
        for (int ri = 0; ri < r; ++ri) {
            for (int si = 0; si < s; ++si) {
                int ihid = ihst + (ri - r / 2) * dh;
                int iwid = iwst + (si - s / 2) * dw;
                if (ihid >= 0 && ihid < h && iwid >= 0 && iwid < w) {
                    imm += *(nfinput + ihid * hchunk + iwid * wchunk + ri * s +
                             si);
                }
            }
        }
        if (bias) {
            imm += bias[fid];
        }
        if (PReLU) {
            imm = imm > 0.0 ? imm : paramReLU * imm;
        }
        output[tid] = imm;
    }
}

__global__ void convTranspose2dreduce_kernel_(
    float *__restrict__ input, float *__restrict__ bias,
    float *__restrict__ output, const bool PReLU, const float paramReLU,
    const int n, const int f, const int h, const int w, const int oh,
    const int ow, const int r, const int s, const int ph, const int pw,
    const int dh, const int dw, const int sh, const int sw) {
    // assert dh = dw = 1
    int nid = blockIdx.x, fid = blockIdx.y;
    int hid = threadIdx.x, wid = threadIdx.y;
    const int fchunck = r * s, wchunk = f * fchunck, hchunk = w * wchunk,
              nchunck = n * hchunk;
    float *nfinput = input + nid * nchunck + fid * fchunck;
    // view as conv, the true ph and pw
    int tph = r - ph - 1, tpw = s - pw - 1;
    int th = (h - 1) * sh + 1, tw = (w - 1) * sw + 1;
    if (nid < n && fid < f && hid < oh && wid < ow) {
        float imm = 0.0;
        int ihst = hid - tph;
        int iwst = wid - tpw;
        for (int ri = 0; ri < r; ++ri) {
            for (int si = 0; si < s; ++si) {
                int ihid = ihst + r - ri - 1;
                int iwid = iwst + s - si - 1;
                if (ihid >= 0 && ihid < th && iwid >= 0 && iwid < tw &&
                    (ihid % sh == 0) && (iwid % sw == 0)) {
                    imm += *(nfinput + (ihid / sh) * hchunk +
                             (iwid / sw) * wchunk + ri * s + si);
                }
            }
        }
        if (bias) {
            imm += bias[fid];
        }
        if (PReLU) {
            imm = imm > 0.0 ? imm : paramReLU * imm;
        }
        output[nid * (oh * ow * f) + hid * (ow * f) + wid * f + fid] = imm;
    }
}

namespace infini {

void conv2dreduce_kernel(float *input, float *bias, float *output, bool PReLU,
                         float paramReLU, int n, int h, int w, int f, int r,
                         int s, int oh, int ow, int ph, int pw, int sh, int sw,
                         int dh, int dw) {
    IT_ASSERT(sh == 1 && sw == 1, "conv2dreduce_kernel only support sh=sw=1");
    const int blocksize = 512;
    const int gridsize = (n * f * oh * ow + blocksize - 1) / blocksize;

    hipStream_t stream(hipStreamPerThread);
    conv2dreduce_kernel_<<<gridsize, blocksize, 0, stream>>>(
        input, bias, output, PReLU, paramReLU, n, f, h, w, oh, ow, r, s, ph, pw,
        dh, dw, sh, sw);
}

void convTranspose2dreduce_kernel(float *input, float *bias, float *output,
                                  bool PReLU, float paramReLU, int n, int h,
                                  int w, int f, int r, int s, int oh, int ow,
                                  int ph, int pw, int sh, int sw, int dh,
                                  int dw) {
    dim3 grid(n, f);
    dim3 block(oh, ow);
    hipStream_t stream(hipStreamPerThread);
    convTranspose2dreduce_kernel_<<<grid, block, 0, stream>>>(
        input, bias, output, PReLU, paramReLU, n, f, h, w, oh, ow, r, s, ph, pw,
        dh, dw, sh, sw);
}
} // namespace infini
