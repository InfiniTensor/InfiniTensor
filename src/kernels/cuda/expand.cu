#include "hip/hip_runtime.h"
#include "core/common.h"
#include "cuda/cuda_common.h"
#include "cuda/cuda_utility.h"
#include "utils/small_array.h"

constexpr unsigned int num_threads() { return 32 * 4; }
constexpr int thread_work_size() { return 4; }
constexpr int block_work_size() { return thread_work_size() * num_threads(); }
const int repeat = 1;
template <class T>
__global__ void _expandKernel(void *input, void *output, int a0, int a1, int a2,
                              int a3, int b0, int b1, int b2, int b3) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int stride1 = b2 * b3;
    int stride0 = b1 * stride1;
    int n = b0 * stride0;
    int end = (repeat * index + repeat < n ? repeat * index + repeat : n);
    for (int i = repeat * index; i < end; i++) {
        int xIdx = (a0 * a1 * a2 * a3 == n ? i : 0);
        bool aIdx = (a0 * a1 * a2 * a3 < n && a0 * a1 * a2 * a3 > 1);
        if (aIdx) {
            int b0_index = i / stride0;
            int b1_index = (i % stride0) / stride1;
            int b2_index = (i % stride1) / b3;
            int b3_index = i % b3;
            xIdx = (b0_index % a0) * a1 * a2 * a3 + (b1_index % a1) * a2 * a3 +
                   (b2_index % a2) * a3 + b3_index % a3;
        }
        ((T *)output)[i] = ((T *)input)[xIdx];
    }
}
template <class T>
__global__ void _expandKernel(void *input, void *output, int nDims,
                              int outputsize, infini::SmallArray inputShape,
                              infini::SmallArray outputShape) {

    int outputIdx =
        blockIdx.x * blockDim.x + threadIdx.x; // i(JKS) + j(KS) + k(S) + s
    if (outputIdx < outputsize) {
        int inputIdx = 0;  // record input index
        int temp = 1;      // stored S, KS, JKS, in order
        int tmp = 1;       // stored s,k,j,i in order
        int v = outputIdx; // v = i(JKS) + j(KS) + k(S) + s
        for (int i = nDims - 1; i >= 0; --i) {
            if (i == 0) {
                tmp = v; // i = outputIdx/(JKS)
            } else {
                tmp = v % outputShape.data[i]; // store s,k,j in order
            }
            if (inputShape.data[i] ==
                1) { // if input shape = 1, the index only equal 0
                inputIdx += 0;
            } else {
                inputIdx +=
                    tmp * temp; // otherwise +i(JKS) or j(KS) or k(S) or s
            }
            temp *= inputShape.data[i];
            v = v / outputShape.data[i];
        }
        ((T *)output)[outputIdx] = ((T *)input)[inputIdx];
    }
}
template <class T>
static __global__ void _expandRowKernel(void *__restrict__ dst,
                                        void const *__restrict__ src) {
    auto da = gridDim.x, db = blockDim.y, dx = blockDim.x, n = blockIdx.y,
         a = blockIdx.x, b = threadIdx.y, x = threadIdx.x;
    auto i = ((n * da + a) * db + b) * dx + x, j = (a * db + b) * dx + x;
    reinterpret_cast<T *>(dst)[i] = reinterpret_cast<T const *>(src)[j];
}
namespace infini {

#define CASE(T)                                                                \
    _expandKernel<DT_CUDA<T>::t>                                               \
        <<<gridsize, blocksize, 0, CUDAStream::getCurrentStream()>>>(          \
            input, output, a0, a1, a2, a3, b0, b1, b2, b3);

#define SWITCH_DTYPE(DTYPE)                                                    \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASE(1)                                                                \
        break;                                                                 \
    case 2:                                                                    \
        CASE(2)                                                                \
        break;                                                                 \
    case 3:                                                                    \
        CASE(3)                                                                \
        break;                                                                 \
    case 4:                                                                    \
        CASE(4)                                                                \
        break;                                                                 \
    case 5:                                                                    \
        CASE(5)                                                                \
        break;                                                                 \
    case 6:                                                                    \
        CASE(6)                                                                \
        break;                                                                 \
    case 7:                                                                    \
        CASE(7)                                                                \
        break;                                                                 \
    case 10:                                                                   \
        CASE(10)                                                               \
        break;                                                                 \
    case 11:                                                                   \
        CASE(11)                                                               \
        break;                                                                 \
    case 12:                                                                   \
        CASE(12)                                                               \
        break;                                                                 \
    case 13:                                                                   \
        CASE(13)                                                               \
        break;                                                                 \
    case 16:                                                                   \
        CASE(16)                                                               \
        break;                                                                 \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }

void expandKernel(int dType, void *input, void *output, int a0, int a1, int a2,
                  int a3, int b0, int b1, int b2, int b3) {
    int blocksize = block_work_size();
    int outputsize = b0 * b1 * b2 * b3;
    int gridsize = (outputsize + repeat * block_work_size() - 1) /
                   (repeat * block_work_size());
    SWITCH_DTYPE(dType)
}
#define CASECurrency(T)                                                        \
    _expandKernel<DT_CUDA<T>::t>                                               \
        <<<gridsize, blocksize, 0, CUDAStream::getCurrentStream()>>>(          \
            input, output, nDims, outputsize, inputShape, outputShape);

#define SWITCHCurrency_DTYPE(DTYPE)                                            \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASECurrency(1) break;                                                 \
    case 2:                                                                    \
        CASECurrency(2) break;                                                 \
    case 3:                                                                    \
        CASECurrency(3) break;                                                 \
    case 4:                                                                    \
        CASECurrency(4) break;                                                 \
    case 5:                                                                    \
        CASECurrency(5) break;                                                 \
    case 6:                                                                    \
        CASECurrency(6) break;                                                 \
    case 7:                                                                    \
        CASECurrency(7) break;                                                 \
    case 10:                                                                   \
        CASECurrency(10) break;                                                \
    case 11:                                                                   \
        CASECurrency(11) break;                                                \
    case 12:                                                                   \
        CASECurrency(12) break;                                                \
    case 13:                                                                   \
        CASECurrency(13) break;                                                \
    case 16:                                                                   \
        CASECurrency(16) break;                                                \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }

void expandKernel(int dType, void *input, void *output, int nDims,
                  int outputsize, SmallArray inputShape,
                  SmallArray outputShape) {
    int blocksize = block_work_size();
    int gridsize = (outputsize + block_work_size() - 1) / block_work_size();
    SWITCHCurrency_DTYPE(dType)
}
#define CASE_ROW(T)                                                            \
    _expandRowKernel<float>                                                    \
        <<<grid, block, 0, CUDAStream::getCurrentStream()>>>(output, input);

#define SWITCH_DTYPE_ROW(DTYPE)                                                \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASE_ROW(1)                                                            \
        break;                                                                 \
    case 2:                                                                    \
        CASE_ROW(2)                                                            \
        break;                                                                 \
    case 3:                                                                    \
        CASE_ROW(3)                                                            \
        break;                                                                 \
    case 4:                                                                    \
        CASE_ROW(4)                                                            \
        break;                                                                 \
    case 5:                                                                    \
        CASE_ROW(5)                                                            \
        break;                                                                 \
    case 6:                                                                    \
        CASE_ROW(6)                                                            \
        break;                                                                 \
    case 7:                                                                    \
        CASE_ROW(7)                                                            \
        break;                                                                 \
    case 10:                                                                   \
        CASE_ROW(10)                                                           \
        break;                                                                 \
    case 11:                                                                   \
        CASE_ROW(11)                                                           \
        break;                                                                 \
    case 12:                                                                   \
        CASE_ROW(12)                                                           \
        break;                                                                 \
    case 13:                                                                   \
        CASE_ROW(13)                                                           \
        break;                                                                 \
    case 16:                                                                   \
        CASE_ROW(16)                                                           \
        break;                                                                 \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }

// Optimization for expanding a row vector. The row length must be a multiple of
// 32
void expandRowKernel(int dType, void *input, void *output, int n_rows,
                     int row_len) {
    // Factorize row_len: row_len = a x b x 32 (32 is the warp size), b<=32
    // input: 1 x (a x b x 32 x sizeT)
    // output: n_rows x (a x b x 32 x sizeT)
    // grid: n_rows x a
    // block: b x 32
    auto c = row_len / 32, b = c;
    if (b > 32) {
        for (b = 32; c % b != 0; --b)
            ;
    }
    auto a = c / b;
    dim3 grid(a, n_rows), block(32, b);
    SWITCH_DTYPE_ROW(dType)
}

} // namespace infini
