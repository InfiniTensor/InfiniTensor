#include "hip/hip_runtime.h"
#include "core/common.h"
#include "cuda/cuda_common.h"
#include "cuda/cuda_utility.h"
#include "utils/small_array.h"

constexpr unsigned int num_threads() { return 32 * 4; }
constexpr int thread_work_size() { return 4; }
constexpr int block_work_size() { return thread_work_size() * num_threads(); }

template <class T>
__global__ void _expandKernel(void *input, void *output, int nDims,
                              int outputsize, infini::SmallArray inputShape,
                              infini::SmallArray outputShape) {

    int outputIdx =
        blockIdx.x * blockDim.x + threadIdx.x; // i(JKS) + j(KS) + k(S) + s
    if (outputIdx < outputsize) {
        int inputIdx = 0;  // record input index
        int temp = 1;      // stored S, KS, JKS, in order
        int tmp = 1;       // stored s,k,j,i in order
        int v = outputIdx; // v = i(JKS) + j(KS) + k(S) + s
        for (int i = nDims - 1; i >= 0; --i) {
            if (i == 0) {
                tmp = v; // i = outputIdx/(JKS)
            } else {
                tmp = v % outputShape.data[i]; // store s,k,j in order
            }
            if (inputShape.data[i] ==
                1) { // if input shape = 1, the index only equal 0
                inputIdx += 0;
            } else {
                inputIdx +=
                    tmp * temp; // otherwise +i(JKS) or j(KS) or k(S) or s
            }
            temp *= inputShape.data[i];
            v = v / outputShape.data[i];
        }
        ((T *)output)[outputIdx] = ((T *)input)[inputIdx];
    }
}

template <class T>
static __global__ void _expandRowKernel(void *__restrict__ dst,
                                        void const *__restrict__ src) {
    auto da = gridDim.x, db = blockDim.y, dx = blockDim.x, n = blockIdx.y,
         a = blockIdx.x, b = threadIdx.y, x = threadIdx.x;
    auto i = ((n * da + a) * db + b) * dx + x, j = (a * db + b) * dx + x;
    reinterpret_cast<T *>(dst)[i] = reinterpret_cast<T const *>(src)[j];
}
namespace infini {

#define CASE(T)                                                                \
    _expandKernel<DT_CUDA<T>::t><<<gridsize, blocksize,                        \
        0, CUDAStream::getCurrentStream()>>>(                                  \
        input, output, nDims, outputsize, inputShape, outputShape);

#define SWITCH_DTYPE(DTYPE)                                                    \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASE(1)                                                                \
        break;                                                                 \
    case 2:                                                                    \
        CASE(2)                                                                \
        break;                                                                 \
    case 3:                                                                    \
        CASE(3)                                                                \
        break;                                                                 \
    case 4:                                                                    \
        CASE(4)                                                                \
        break;                                                                 \
    case 5:                                                                    \
        CASE(5)                                                                \
        break;                                                                 \
    case 6:                                                                    \
        CASE(6)                                                                \
        break;                                                                 \
    case 7:                                                                    \
        CASE(7)                                                                \
        break;                                                                 \
    case 10:                                                                   \
        CASE(10)                                                               \
        break;                                                                 \
    case 11:                                                                   \
        CASE(11)                                                               \
        break;                                                                 \
    case 12:                                                                   \
        CASE(12)                                                               \
        break;                                                                 \
    case 13:                                                                   \
        CASE(13)                                                               \
        break;                                                                 \
    case 16:                                                                   \
        CASE(16)                                                               \
        break;                                                                 \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }

void expandKernel(int dType, void *input, void *output, int nDims,
                  int outputsize, SmallArray inputShape,
                  SmallArray outputShape) {
    int blocksize = block_work_size();
    int gridsize = (outputsize + block_work_size() - 1) / block_work_size();
    SWITCH_DTYPE(dType)
}

#define CASE_ROW(T)                                                            \
    _expandRowKernel<float>                                                    \
        <<<grid, block, 0, CUDAStream::getCurrentStream()>>>(output, input);

#define SWITCH_DTYPE_ROW(DTYPE)                                                \
    switch (DTYPE) {                                                           \
    case 1:                                                                    \
        CASE_ROW(1)                                                            \
        break;                                                                 \
    case 2:                                                                    \
        CASE_ROW(2)                                                            \
        break;                                                                 \
    case 3:                                                                    \
        CASE_ROW(3)                                                            \
        break;                                                                 \
    case 4:                                                                    \
        CASE_ROW(4)                                                            \
        break;                                                                 \
    case 5:                                                                    \
        CASE_ROW(5)                                                            \
        break;                                                                 \
    case 6:                                                                    \
        CASE_ROW(6)                                                            \
        break;                                                                 \
    case 7:                                                                    \
        CASE_ROW(7)                                                            \
        break;                                                                 \
    case 10:                                                                   \
        CASE_ROW(10)                                                           \
        break;                                                                 \
    case 11:                                                                   \
        CASE_ROW(11)                                                           \
        break;                                                                 \
    case 12:                                                                   \
        CASE_ROW(12)                                                           \
        break;                                                                 \
    case 13:                                                                   \
        CASE_ROW(13)                                                           \
        break;                                                                 \
    case 16:                                                                   \
        CASE_ROW(16)                                                           \
        break;                                                                 \
    default:                                                                   \
        IT_TODO_HALT();                                                        \
    }

// Optimization for expanding a row vector. The row length must be a multiple of 32
void expandRowKernel(int dType, void *input, void *output, int n_rows,
                     int row_len) {
    // Factorize row_len: row_len = a x b x 32 (32 is the warp size), b<=32
    // input: 1 x (a x b x 32 x sizeT)
    // output: n_rows x (a x b x 32 x sizeT)
    // grid: n_rows x a
    // block: b x 32
    auto c = row_len / 32, b = c;
    if (b > 32) {
        for (b = 32; c % b != 0; --b);
    }
    auto a = c / b;
    dim3 grid(a, n_rows), block(32, b);
    SWITCH_DTYPE_ROW(dType)
}

} // namespace infini
