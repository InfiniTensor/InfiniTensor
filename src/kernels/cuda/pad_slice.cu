#include "hip/hip_runtime.h"
#include "cuda/cuda_common.h"
#include "cuda/cuda_pad_slice.h"

__device__ int WholeTensorOffset2PartTensorOffset(int wholeOffset,
                                                  TransMetaData metaData,
                                                  int nDims) {
    int offset = 0;
    for (int i = nDims - 1; i >= 0; --i) {
        auto wholePos = wholeOffset % metaData.wholeNDim[i];
        auto pos = wholePos - metaData.begNum[i];
        // if pos belongs to pad range, then return -1
        if (pos < 0 || pos >= metaData.partNDim[i])
            return -1;
        wholeOffset = wholeOffset / metaData.wholeNDim[i];

        offset += pos * metaData.partStride[i];
    }

    return offset;
}

__global__ void _pad_slice_kernel(float *part, float *whole,
                                  TransMetaData metaData, int nDims, int num,
                                  bool isPad) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num)
        return;

    int stride = blockDim.x * gridDim.x;
    while (tid < num) {
        int offset = WholeTensorOffset2PartTensorOffset(tid, metaData, nDims);
        if (isPad)
            if (offset < 0)
                whole[tid] = 0;
            else
                whole[tid] = part[offset];
        else
            part[offset] = whole[tid];
        tid += stride;
    }
}

namespace infini {
void pad_slice_kernel(float *partData, float *wholeData,
                      const TransMetaData &metadata, int nDims, int num,
                      bool isPad) {
    int blockSize = 32 * 16;
    int gridSize = (num + blockSize - 1) / blockSize;
    _pad_slice_kernel<<<gridSize, blockSize>>>(partData, wholeData, metadata,
                                               nDims, num, isPad);
}
} // namespace infini
