#include "hip/hip_runtime.h"
#include "cuda_utils.h"
// Kernel
__global__ void kernel_func_0(float *tensor_ptr_2, float *tensor_ptr_3) {
    int lane_id = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;
    int parallel_idx = blockIdx.x * 8 + warp_id;
    float buf[8];
    for (int loop_idx = parallel_idx; loop_idx < 65536; loop_idx += 864) {
        int offset_src = 0;
        int offset_src_buf = loop_idx;
        offset_src += offset_src_buf % 65536 * 256;
        offset_src_buf /= 65536;
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx] =
                tensor_ptr_2[0 + offset_src + inst_idx * 32 + lane_id];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx] = (buf[inst_idx] > 0) ? buf[inst_idx] : 0;
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            tensor_ptr_3[0 + offset_src + inst_idx * 32 + lane_id] =
                buf[inst_idx];
        }
    }
}
// Kernel
__global__ void kernel_func_1(float *tensor_ptr_2, float *tensor_ptr_3,
                              float *tensor_ptr_4) {
    int lane_id = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;
    int parallel_idx = blockIdx.x * 8 + warp_id;
    float buf[24];
    for (int loop_idx = parallel_idx; loop_idx < 65536; loop_idx += 864) {
        int offset_src = 0;
        int offset_src_buf = loop_idx;
        offset_src += offset_src_buf % 65536 * 256;
        offset_src_buf /= 65536;
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx] =
                tensor_ptr_2[0 + offset_src + inst_idx * 32 + lane_id];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx + 8] =
                tensor_ptr_3[0 + offset_src + inst_idx * 32 + lane_id];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx + 16] = buf[inst_idx] + buf[inst_idx + 8];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            tensor_ptr_4[0 + offset_src + inst_idx * 32 + lane_id] =
                buf[inst_idx + 16];
        }
    }
}
void invoke_func_0(float *tensor_ptr_2, float *tensor_ptr_3) {
    dim3 gridDim(108, 1);
    dim3 blockDim(256, 1);
    kernel_func_0<<<gridDim, blockDim>>>(tensor_ptr_2, tensor_ptr_3);
    cudaCheckError();
}
void invoke_func_1(float *tensor_ptr_2, float *tensor_ptr_3,
                   float *tensor_ptr_4) {
    dim3 gridDim(108, 1);
    dim3 blockDim(256, 1);
    kernel_func_1<<<gridDim, blockDim>>>(tensor_ptr_2, tensor_ptr_3,
                                         tensor_ptr_4);
    cudaCheckError();
}
