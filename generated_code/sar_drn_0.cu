#include "hip/hip_runtime.h"
#include "cuda_utils.h"
// Kernel
__global__ void kernel_func_2(float *tensor_ptr_2, float *tensor_ptr_4,
                              float *tensor_ptr_5) {
    int lane_id = threadIdx.x % 32;
    int warp_id = threadIdx.x / 32;
    int parallel_idx = blockIdx.x * 8 + warp_id;
    float buf[32];
    for (int loop_idx = parallel_idx; loop_idx < 65536; loop_idx += 864) {
        int offset_src = 0;
        int offset_src_buf = loop_idx;
        offset_src += offset_src_buf % 65536 * 256;
        offset_src_buf /= 65536;
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx] =
                tensor_ptr_2[0 + offset_src + inst_idx * 32 + lane_id];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx] = (buf[inst_idx] > 0) ? buf[inst_idx] : 0;
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx + 8] =
                tensor_ptr_4[0 + offset_src + inst_idx * 32 + lane_id];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            buf[inst_idx + 16] = buf[inst_idx] + buf[inst_idx + 8];
        }
#pragma unroll
        for (int inst_idx = 0; inst_idx < 8; inst_idx++) {
            tensor_ptr_5[0 + offset_src + inst_idx * 32 + lane_id] =
                buf[inst_idx + 16];
        }
    }
}
void invoke_func_2(float *tensor_ptr_2, float *tensor_ptr_4,
                   float *tensor_ptr_5) {
    dim3 gridDim(108, 1);
    dim3 blockDim(256, 1);
    kernel_func_2<<<gridDim, blockDim>>>(tensor_ptr_2, tensor_ptr_4,
                                         tensor_ptr_5);
    cudaCheckError();
}
